#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <bits/stdc++.h>

using namespace std;

// Number of variables (observations).  Equal to codeword length, n.
#define N_VARIABLES 10
// Number of edges to/from each variable (also the number of 1's in each col of H)
#define D_V 2

// Number of factors (parity checks).  Equal to parity length, k.
#define N_FACTORS 5
// Number of edges to/from each factor (also the number of 1's in each row of H)
#define D_C 4

// Channel Models (determines probability algorithm)
#define ALGO_SBC 1
#define ALGO_SEC 2
#define ALGO_AWGN 3

struct ldpc_info_t {
    bool valid;
    int iterations;
    int max_iterations;
};

__device__ void update_variable(int* f_indexes, int* v_indexes, float* msg_v_to_f, float* msg_f_to_v, float *observations, int v) {
    // update outgoing messages from a single variable to all factors
    if (v < N_VARIABLES) {
        // intitially sum = observation
        float sum = observations[v];
        // iteration 1: calculate sum over all contributing factors
        const int offset = v * D_V;
        for (int f=0; f < D_V; f++) {
            const int mi = v_indexes[offset + f];
            if (mi == -1) break;
            sum += msg_f_to_v[mi];
        }
        // iteration 2: subtract incoming contribution from outgoing edge
        for (int f=0; f < D_V; f++) {
            const int mi = v_indexes[offset + f];
            if (mi == -1) break;
            msg_v_to_f[v * D_V + f] = sum - msg_f_to_v[mi];
        }
    }
}

__device__ float arctanhf(float x) {
    return 0.5f * logf((1.0f + x) / (1.0f - x));
}

__device__ void update_factor(int* f_indexes, int* v_indexes, float* msg_v_to_f, float* msg_f_to_v, int f) {
    // update outgoing messages from a single factor to all variables
    if (f < N_FACTORS) {
        float L=1;
        
        // iteration 1: calculate L over all contributing variables
        const int offset = f * D_C;
        for (int v=0; v < D_C; v++) {
            const int mi = f_indexes[offset + v];
            if (mi == -1) break;
            L *= tanhf(msg_v_to_f[mi]/2.0f);
        }
        // iteration 2: divide (remove) incoming contribution from outgoing edge
        for (int v=0; v < D_C; v++) {
            const int mi = f_indexes[offset + v];
            if (mi == -1) break;
            msg_f_to_v[f*D_C + v] = 2.0f*arctanhf(L / tanhf(msg_v_to_f[mi]/2.0f));
        }
    }
}

__device__ void update_beliefs(int* f_indexes, int* v_indexes, float* msg_v_to_f, float* msg_f_to_v, float *observations, float *beliefs) {
    // like update_variable, but doesn't subtract any incoming messages
    for (int v = 0; v < N_VARIABLES; v++) {
        float sum = observations[v];

        // calculate sum over all contributing factors
        const int offset = v * D_V;
        for (int f=0; f < D_V; f++) {
            const int mi = v_indexes[offset + f];
            if (mi == -1) break;
            sum += msg_f_to_v[mi];
        }
        
        beliefs[v] = sum;
    }
}

__device__ bool valid_codeword(int* f_indexes, int* v_indexes, float *observations, float *beliefs) {
    
    for (int f=0; f < N_FACTORS; f++) {
        int parity_check = 0;
        const int offset = f * D_C;
        for (int v=0; v < D_C; v++) {
            int mi = f_indexes[offset + v];
            if (mi == -1) break;
            mi /= D_V;

            // observations are positive/negative for bit values of 0/1, respectively (yes, it seems backwards)
            // beliefs are negative/positive for disbelief/belief in the value of the observation
            //    ergo, beliefs multiplied by observations will simply invert the logic
            parity_check ^= (beliefs[mi] < 0) ? 1 : 0;
        }
        if (parity_check != 0) return false;

    }
    
    return true;
}

// CUDA kernel: add row index to each element in the row
__global__ void iterateLDPC(int* f_indexes, int* v_indexes, float* msg_v_to_f, float* msg_f_to_v, float *observations, float *beliefs, ldpc_info_t *ldpc_info) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ bool codeword_is_valid;
    
    if (x >= N_VARIABLES) return;
    
    for (int i=0; i < ldpc_info->max_iterations; i++) {
        update_variable(f_indexes, v_indexes, msg_v_to_f, msg_f_to_v, observations, x);

        __syncthreads(); // sync required; factors require multiple variables

        update_factor(f_indexes, v_indexes, msg_v_to_f, msg_f_to_v, x);

        // Early iteration break check
        if (threadIdx.x == 0) {
            update_beliefs(f_indexes, v_indexes, msg_v_to_f, msg_f_to_v, observations, beliefs);
            codeword_is_valid = valid_codeword(f_indexes, v_indexes, observations, beliefs);
            ldpc_info->iterations = i + 1;
            ldpc_info->valid = codeword_is_valid;
        }

        __syncthreads();
        
        
        if (codeword_is_valid) {
            break;
        }
    }
}

void map_h_to_indexes(int *h_matrix, int *f_indexes, int *v_indexes) {
    // create mappings of edges from factors and variables, indexed into each other
    for (int f=0; f < N_FACTORS; f++) {
        int fi = 0;
        for (int v=0; v < N_VARIABLES; v++) {
            if (h_matrix[f * N_VARIABLES + v]) {

                int vi=0;
                while (v_indexes[(v * D_V + vi)] != -1) {
                    vi++;
                }
                v_indexes[(v * D_V + vi)] = f * D_C + fi;
                f_indexes[(f * D_C + fi)] = v * D_V + vi;

                fi++;
            }
        }
    }
}

int h_matrix[N_VARIABLES * N_FACTORS] =
    {1, 1, 1, 1, 0, 0, 0, 0, 0, 0,
     0, 1, 0, 0, 1, 0, 0, 0, 1, 1,
     1, 0, 1, 0, 0, 1, 1, 0, 0, 0,
     0, 0, 0, 1, 1, 0, 1, 1, 0, 0,
     0, 0, 0, 0, 0, 1, 0, 1, 1, 1};

                               // {1, 0, 1, 0, 1, 1, 1, 0, 1, 0}; // actual valid codeword
//float observations[N_VARIABLES] = {1, 1, 1, 0, 1, 1, 1, 0, 1, 0}; // hard erroneous codeword
float observations[N_VARIABLES] = {0.9, 0.1, 0.4, -0.7, 0.3, 0.8, 0.945, -0.6, 0.5, -0.76}; // soft erroneous codeword


int v_indexes[N_VARIABLES * D_V]; // variable indexes (that is, which factors are used to make a variable)
int f_indexes[N_FACTORS * D_C]; // factor indexes (that is, which variables are connected to a factor)

float messages_v_to_f[N_VARIABLES * D_V];
float messages_f_to_v[N_FACTORS * D_C];

float beliefs[N_VARIABLES];
int iterations = 10;


int main() {
    ldpc_info_t ldpc_info;
    ldpc_info.max_iterations = 10;

    hipDeviceProp_t prop;
    int device = 0;
    hipGetDeviceProperties(&prop, device);
    int maxThreadsPerBlock = prop.maxThreadsPerBlock;
    // Fallback for compute capability 6.1 (Pascal, e.g., GTX 1080): 1024 threads per block
    if (maxThreadsPerBlock <= 0) maxThreadsPerBlock = 1024;
    
    // Map the matrix to indexes
    memset(v_indexes, -1, sizeof(v_indexes));
    memset(f_indexes, -1, sizeof(f_indexes));
    map_h_to_indexes(h_matrix, f_indexes, v_indexes);
    
    // Output matrix and original observations
    std::cout << "Original matrix:\n";
    for (int f = 0; f < N_FACTORS; f++) {
        for (int v = 0; v < N_VARIABLES; ++v)
            std::cout << h_matrix[f*N_VARIABLES+v] << " ";
        std::cout << "\n";
    }
    std::cout << "\nObservations In:\n";
    for (int i = 0; i < N_VARIABLES; ++i) {
        std::cout << observations[i] << " ";
    }
    std::cout << "\n";

    // Update observations -> LLR
    for (int i=0; i < N_VARIABLES; ++i) {
        observations[i] = -2*observations[i];
    }
    
    // Allocate device memory
    int *cuda_f_indexes = NULL, *cuda_v_indexes = NULL;
    hipMalloc(&cuda_f_indexes, N_FACTORS * D_C * sizeof(int));
    hipMalloc(&cuda_v_indexes, N_VARIABLES * D_V * sizeof(int));

    float *cuda_observations = NULL, *cuda_beliefs = NULL;
    hipMalloc(&cuda_observations, N_VARIABLES * sizeof(float));
    hipMalloc(&cuda_beliefs, N_VARIABLES * sizeof(float)); // extra floats: [0] 0=invalid, 1=valid, [1] # iter
    
    ldpc_info_t *cuda_ldpc_info;
    hipMalloc(&cuda_ldpc_info, sizeof(ldpc_info_t));

    float *cuda_messages_v_to_f = NULL, *cuda_messages_f_to_v = NULL;
    hipMalloc(&cuda_messages_v_to_f, N_VARIABLES * D_V * sizeof(float));
    hipMalloc(&cuda_messages_f_to_v, N_FACTORS * D_C * sizeof(float));

    // Copy matrix to device
    hipMemcpy(cuda_f_indexes, f_indexes, N_FACTORS * D_C * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(cuda_v_indexes, v_indexes, N_VARIABLES * D_V * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(cuda_observations, observations, N_VARIABLES * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(cuda_ldpc_info, &ldpc_info, sizeof(ldpc_info_t), hipMemcpyHostToDevice);

    // Launch kernel: one thread per row
    int threadsPerBlock = maxThreadsPerBlock;
    int blocksPerGrid = (N_VARIABLES + threadsPerBlock - 1) / threadsPerBlock;

    iterateLDPC<<<blocksPerGrid, threadsPerBlock>>>(cuda_f_indexes, cuda_v_indexes, cuda_messages_v_to_f, cuda_messages_f_to_v, cuda_observations, cuda_beliefs, cuda_ldpc_info);
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(observations, cuda_observations, N_VARIABLES * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(beliefs, cuda_beliefs, N_VARIABLES * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&ldpc_info, cuda_ldpc_info, sizeof(ldpc_info_t), hipMemcpyDeviceToHost);

    printf("Valid = %d  Iterations Calculated = %d\n", ldpc_info.valid, ldpc_info.iterations);
    
    printf("  Original observations:   ");
    for (int v=0; v < N_VARIABLES; v++) {
        printf("%d", (observations[v] < 0) ? 1 : 0);
    }
    printf("\n");
    printf("  Final hard codeword found as: ");
    for (int v=0; v < N_VARIABLES; v++) {
        printf("%d", (beliefs[v] < 0) ? 1 : 0);
    }
    printf("\n");
    printf("  Final soft codeword found as: ");
    for (int v=0; v < N_VARIABLES; v++) {
        printf("%f ", beliefs[v] / -2);
    }
    printf("\n");
    printf("  Codeword stats: ");
    for (int v=0; v < N_VARIABLES; v++) {
        printf("%d (o=%f b=%f) ", (beliefs[v] < 0) ? 1 : 0, observations[v], beliefs[v]);
    }
    printf("\n");

    // Free CUDA memory
    hipFree(cuda_f_indexes);
    hipFree(cuda_v_indexes);
    hipFree(cuda_observations);
    hipFree(cuda_beliefs);
    hipFree(cuda_ldpc_info);
    hipFree(cuda_messages_v_to_f);
    hipFree(cuda_messages_f_to_v);

    return 0;
}

